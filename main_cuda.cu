
#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MIN(a, b) (((a) < (b)) ? (a) : (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))

double vetor[16777216];
double vetor_log[16777216];
int n_threads;

void swap(double *a, double *b) {
    double t = *a;
    *a = *b;
    *b = t;
}

int divide(double arr[], int low, int high) {
    double pivot = arr[high];
    int i = (low - 1);

    for (int j = low; j <= high - 1; j++)
        if (arr[j] < pivot) swap(&arr[++i], &arr[j]);
    swap(&arr[i + 1], &arr[high]);
    return (i + 1);
}

void quicksort(double arr[], int low, int high) {
    if (low < high) {
        int pivot = divide(arr, low, high);
#pragma omp task final((pivot - low) < n_threads) mergeable default(none) \
    shared(arr) firstprivate(low, pivot)
        quicksort(arr, low, pivot - 1);

#pragma omp task final((high - pivot) < n_threads) mergeable default(none) \
    shared(arr) firstprivate(high, pivot)
        quicksort(arr, pivot + 1, high);
    }
}

int open_file(int idx, char *files[]) {
    FILE *filep;
    char filename[100];
    strcpy(filename, "entradas/");
    strcat(filename, files[idx]);
    filep = fopen(filename, "r");
    int index = 0;
    if (filep == NULL) {
        fprintf(stderr, "Cannot open file %s\n", filename);
        exit(1);
    } else {
        while ((fscanf(filep, "%lf", &vetor[index++])) != EOF) {
            vetor_log[index - 1] = log(vetor[index - 1]);
        }
    }
    fclose(filep);
    return index - 1;
}

// cuda sum
static const int blockSize = 1024;
static const int gridSize = 24;

__device__ double sum_warp(volatile double *shArr) {
    int idx = threadIdx.x % warpSize;  // quantidade de wrap permitido pela GPU
    if (idx < 16) {
        shArr[idx] += shArr[idx + 16];
        shArr[idx] += shArr[idx + 8];
        shArr[idx] += shArr[idx + 4];
        shArr[idx] += shArr[idx + 2];
        shArr[idx] += shArr[idx + 1];
    }
    return shArr[0];
}

__global__ void sum_reduction(const double *a, const double arraySize,
                              double *out) {
    int idx = threadIdx.x;  // idx do bloco
    double sum = 0;
#pragma omp parallel for reduction(+ : sum)
    for (int i = idx; i < arraySize; i += blockSize) sum += a[i];
    __shared__ double r[blockSize];
    r[idx] = sum;
    sum_warp(&r[idx & ~(warpSize - 1)]);  // soma dos valores do bloco
    __syncthreads();                      // barreira
    if (idx < warpSize) {
        r[idx] = idx * warpSize < blockSize ? r[idx * warpSize]
                                            : 0;  // lidando com overflow
        sum_warp(r);                              // computando bloco do inicio
        if (idx == 0) *out = r[0];  // salvando na posicao 0 a soma
    }
}

__host__ double sumArray(double *arr, int size) {
    double *dev_arr;
    hipMalloc((void **)&dev_arr, size * sizeof(double));
    hipMemcpy(dev_arr, arr, size * sizeof(double), hipMemcpyHostToDevice);

    double out;
    double *dev_out;
    hipMalloc((void **)&dev_out, sizeof(double) * gridSize);

    int *dev_lastBlockCounter;
    hipMalloc((void **)&dev_lastBlockCounter, sizeof(int));
    hipMemset(dev_lastBlockCounter, 0, sizeof(int));

    sum_reduction<<<gridSize, blockSize>>>(dev_arr, size, dev_out);
    hipDeviceSynchronize();  // barreira

    hipMemcpy(&out, dev_out, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(dev_arr);
    hipFree(dev_out);
    return out;
}

void solve(int idx, char *files[], int size) {
    // sorting
#pragma omp parallel
#pragma omp single
    quicksort(vetor, 0, size - 1);
    // ===============

    // aritmetical and geometric mean
    double ma = 0, mg = 0;
    ma = sumArray(vetor, size) / (double)size;
    mg = sumArray(vetor_log, size) / (double)size;
    mg = exp(mg);
    // ===============

    // positional values
    double md = vetor[size / 2 - 1];
    double p95 = vetor[(95 * (size + 1)) / 100 - 1];
    double maximo = vetor[size - 1];
    double minimo = vetor[0];
    // ===============

    double aux_dp = 0.0;
    int i;
#pragma omp parallel for reduction(+ : aux_dp) schedule(dynamic) \
    private(i) shared(vetor, size)
    for (i = 0; i < size; i++) {
        double aux_x = vetor[i] - ma;
        aux_dp += (aux_x * aux_x);
    }
    double dp = sqrt(aux_dp / size);

    FILE *filep;
    filep = fopen("saidas/results", "a");
    fprintf(filep, "%s %.6lf %.6lf %.6lf %.6lf %.6lf %.6lf %.6lf\n", files[idx],
            ma, dp, mg, md, p95, minimo, maximo);
    fclose(filep);
}

int main(int argc, char *argv[]) {
    int i, n;

    n_threads = omp_get_max_threads();
    omp_set_num_threads(n_threads);

    double start_time_total = omp_get_wtime();

    for (i = 1; i < argc; i++) {
        n = open_file(i, argv);
        double start_time = omp_get_wtime();
        solve(i, argv, n);
        double end_time = omp_get_wtime();
        printf("%s - tempo: %.6lfs\n", argv[i], end_time - start_time);
    }

    double end_time_total = omp_get_wtime();

    printf("Tempo total: %.6lfs\n", end_time_total - start_time_total);

    return 0;
}
